#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""

#include <stdio.h>      
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include<stdio.h>

#define COUNT 7
#define ROWS 12

struct returnObject{
	int omega[COUNT];
	int o[COUNT];
};

struct vertex{
	int x, y;
};


__device__ int findQueueSize(){
	return 0;
}

int findQueueSiz(){
	return 0;
}
struct returnObject *globalQueue;
__device__ void enqueue(struct returnObject *globalQueue, int *omega, int *o){
	int i;
	for (i = 0; i<COUNT; i++){
		globalQueue[findQueueSize()].omega[i] = omega[i];
		globalQueue[findQueueSize()].o[i] = o[i];
	}
}


__device__ int findSize(int a[]){
	int i = 0;
	int counter = 0;
	for (i = 0; i<COUNT; i++)
	if (a[i] == 1)
		counter++;
	return (counter);
}

__device__ int findGraph(struct vertex *a){
	//int i=0;
	//while(i<ROWS && a[i++][0]!=-1);
	return COUNT;
}

__device__ int lastBit(int a[]){
	int i;
	for (i = 0; i<COUNT; i++)
	if (a[i] == -1)
		return i;
	return i;
}

__device__ int firstSetBit(int a[]){
	int i;
	for (i = 0; i<COUNT; i++)
	if (a[i] == 1)
		return i;
	return i;
}


__device__ struct returnObject *colourise(struct vertex *graph, int* adjacentVertices){
	struct returnObject *returnobject = (returnObject *)malloc(sizeof(struct returnObject));
	int i;
	for (i = 0; i<COUNT; i++){
		returnobject->omega[i] = -1;
		returnobject->o[i] = -1;
	}
	int pDash[COUNT];
	int j = 0;
	for (i = 0; i<COUNT; i++){
		pDash[i] = adjacentVertices[i];
	}
	int om = 0;
	while (findSize(pDash)>0){
		om++;
		int q[COUNT];
		for (j = 0; j<COUNT; j++){
			q[j] = pDash[j];
		}
		while (findSize(q)>0){
			int v = firstSetBit(q);
			pDash[v] = -1;
			q[v] = -1;
			int neighbours[COUNT];
			for (j = 0; j<COUNT; j++)
				neighbours[j] = -1;
			int graphSize = findGraph(graph);
			for (j = 0; j <= graphSize; j++){
				if (graph[j].x == v){
					neighbours[graph[j].y] = 1;
				}
				else if (graph[j].y == v){
					neighbours[graph[j].x] = 1;
				}
			}
			for (j = 0; j<COUNT; j++)
			if (j != v && (neighbours[j] == 0 || neighbours[j] == -1))
				neighbours[j] = 1;
			else
				neighbours[j] = -1;
			for (j = 0; j<COUNT; j++){
				if (q[j] == neighbours[j] && q[j] == 1)
					q[j] = 1;
				else
					q[j] = -1;
			}
			returnobject->omega[lastBit(returnobject->omega)] = om;
			returnobject->o[lastBit(returnobject->o)] = v;
		}
	}
	return returnobject;
}


__device__ void expand1(struct vertex *graph, int* candidateClique, int* adjacentVertices, int* incumbentVertices,struct returnObject *globalQueue,int populate){
	//	struct returnObject *returnobject1=malloc(sizeof(struct returnObject));
	int i; 
	
	
	struct returnObject *returnobject = (returnObject *)malloc(sizeof(struct returnObject));
	
	for (i = 0; i<COUNT; i++){
		returnobject->omega[i] = -1;
		returnobject->o[i] = -1;
	}
	returnobject = colourise(graph, adjacentVertices);
	//	hipMemcpy(returnobject,returnobject1,sizeof(returnObject),hipMemcpyDeviceToDevice);
	int adjacentSize = findSize(adjacentVertices);
	for (i = adjacentSize - 1; i >= 0; i--){
		int candidateSize = findSize(candidateClique);
		int incumbentSize = findSize(incumbentVertices);
		if (candidateSize + returnobject->omega[i]>incumbentSize){
			int v = returnobject->o[i];
			candidateClique[v] = 1;
			int neighbours[10] = { -1 };
			int graphSize = findGraph(graph);
			int j = 0;
			for (j = 0; j <= graphSize; j++){
				if (graph[j].x == v){
					neighbours[graph[j].y] = 1;
				}
				else if (graph[j].y == v){
					neighbours[graph[j].x] = 1;
				}
			}
			int *newAdjacent = (int *)malloc(sizeof(int)*COUNT);
			for (j = 0; j<COUNT; j++)
				newAdjacent[j] = -1;
			for (j = 0; j<COUNT; j++)
			if (adjacentVertices[j] == neighbours[j] && neighbours[j] == 1)
				newAdjacent[j] = 1;
			int newAdjacentSize = findSize(newAdjacent);
			if (newAdjacentSize == 0){
				candidateSize = findSize(candidateClique);
				incumbentSize = findSize(incumbentVertices);
				if (candidateSize>incumbentSize){
					for (j = 0; j<COUNT; j++){
						incumbentVertices[j] = candidateClique[j];
					}
				}
			}
			else{
				if (populate == 1)
					enqueue(globalQueue, candidateClique, newAdjacent);
				else
					expand1(graph, candidateClique, newAdjacent, incumbentVertices,globalQueue,populate);
			}
			candidateClique[v] = -1;
			adjacentVertices[v] = -1;
		}
	}
}


int globalIncumbent[10][10] = { -1 };
int globalCounter = 0;

void newGlobal(int a[]){
	int i, j;
	globalCounter = 0;
	for (i = 0; i<COUNT; i++)
	for (j = 0; j<COUNT; j++)
		globalIncumbent[i][j] = -1;
	for (i = 0; i<COUNT; i++)
		globalIncumbent[globalCounter][i] = a[i];
	globalCounter++;
}

void appendGlobal(int a[]){
	int i;
	for (i = 0; i<COUNT; i++)
		globalIncumbent[globalCounter][i] = a[i];
	globalCounter++;
}

int findSize1(int a[]){
	int i = 0;
	int counter = 0;
	for (i = 0; i<COUNT; i++)
	if (a[i] == 1)
		counter++;
	return (counter);
}

int findGraph1(int a[10][2]){
	int i = 0;
	while (i<COUNT && a[i++][0] != -1);
	return i - 1;
}

int lastBit1(int a[]){
	int i;
	for (i = 0; i<COUNT; i++)
	if (a[i] == -1)
		return i;
}

int firstSetBit1(int a[]){
	int i;
	for (i = 0; i<COUNT; i++)
	if (a[i] == 1)
		return i;
}
struct returnObject *colourise(int graph[10][2], int* adjacentVertices){
	struct returnObject *returnobject =(struct returnObject*) malloc(sizeof(struct returnObject));
	int i;
	for (i = 0; i<COUNT; i++){
		returnobject->omega[i] = -1;
		returnobject->o[i] = -1;
	}
	int pDash[10] = { -1 };
	int j = 0, k = 0;
	for (i = 0; i<COUNT; i++){
		pDash[i] = adjacentVertices[i];
	}
	int om = 0;
	while (findSize1(pDash)>0){
		om++;
		int q[10] = { -1 };
		for (j = 0; j<COUNT; j++){
			q[j] = pDash[j];
		}
		while (findSize1(q)>0){
			int v = firstSetBit1(q);
			pDash[v] = -1;
			q[v] = -1;
			int neighbours[10] = { -1 };
			int graphSize = findGraph1(graph);
			k = 0;
			for (j = 0; j <= graphSize; j++){
				if (graph[j][0] == v){
					neighbours[graph[j][1]] = 1;
				}
				else if (graph[j][1] == v){
					neighbours[graph[j][0]] = 1;
				}
			}
			for (j = 0; j<COUNT; j++)
			if (j != v && (neighbours[j] == 0 || neighbours[j] == -1))
				neighbours[j] = 1;
			else
				neighbours[j] = -1;
			for (j = 0; j<COUNT; j++){
				if (q[j] == neighbours[j] && q[j] == 1)
					q[j] = 1;
				else
					q[j] = -1;
			}
			returnobject->omega[lastBit1(returnobject->omega)] = om;
			returnobject->o[lastBit1(returnobject->o)] = v;
		}
	}
	//	printf("about to return!!\n");
	return returnobject;
}


void expand(int graph[10][2], int* candidateClique, int* adjacentVertices, int* incumbentVertices){
	struct returnObject *returnobject =(struct returnObject*)malloc(sizeof(struct returnObject));
	int i;
	//	printf("adjacentVertices:");
	for (i = 0; i<COUNT; i++){
		//		printf("%d   ",adjacentVertices[i]);
		returnobject->omega[i] = -1;
		returnobject->o[i] = -1;
	}
	/*	printf("\ncandidateClique:");
	for (i = 0; i < COUNT; ++i){
	printf("%d   ",candidateClique[i]);
	}
	printf("\nincumbentVertices:");
	for (i = 0; i < COUNT; ++i){
	printf("%d   ",incumbentVertices[i]);
	}*/
	//	printf("\ncolourise\n");
	returnobject = colourise(graph, adjacentVertices);
	/*	for(i=0;i<COUNT;i++){
	printf("%d   ",returnobject->omega[i]);
	}
	printf("\n");
	for(i=0;i<COUNT;i++){
	printf("%d   ",returnobject->o[i]);
	}*/
	//	printf("\nexpand\n");
	int adjacentSize = findSize1(adjacentVertices);
	for (i = adjacentSize - 1; i >= 0; i--){
		int candidateSize = findSize1(candidateClique);
		int incumbentSize = findSize1(incumbentVertices);
		if (candidateSize + returnobject->omega[i]>incumbentSize){
			int v = returnobject->o[i];
			candidateClique[v] = 1;
			int neighbours[10] = { -1 };
			int graphSize = findGraph1(graph);
			int j = 0;
			for (j = 0; j <= graphSize; j++){
				if (graph[j][0] == v){
					neighbours[graph[j][1]] = 1;
				}
				else if (graph[j][1] == v){
					neighbours[graph[j][0]] = 1;
				}
			}
			int *newAdjacent = (int*)malloc(sizeof(int)* 10);
			for (j = 0; j<COUNT; j++)
				newAdjacent[j] = -1;
			for (j = 0; j<COUNT; j++)
			if (adjacentVertices[j] == neighbours[j] && neighbours[j] == 1)
				newAdjacent[j] = 1;
			int newAdjacentSize = findSize1(newAdjacent);
			if (newAdjacentSize == 0){
				candidateSize = findSize1(candidateClique);
				incumbentSize = findSize1(incumbentVertices);
				if (candidateSize>incumbentSize){
					newGlobal(candidateClique);
					for (j = 0; j<COUNT; j++){
						incumbentVertices[j] = candidateClique[j];
					}
				}
				else if (candidateSize == incumbentSize){
					appendGlobal(candidateClique);
				}
			}
			else{
				expand(graph, candidateClique, newAdjacent, incumbentVertices);
			}
			candidateClique[v] = -1;
			adjacentVertices[v] = -1;
		}
	}
}

__global__ void expand(struct vertex *graph, int* candidateClique, int* adjacentVertices, int* incumbentVertices,struct returnObject *globalQueue,int populate){
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	if (populate == 0){
		int index = threadIdx.x + blockIdx.x*blockDim.x;
		candidateClique = globalQueue[index].omega;
		adjacentVertices = globalQueue[index].o;
		expand1(graph, candidateClique, adjacentVertices, incumbentVertices, globalQueue, populate);

	}
	else{
		expand1(graph, candidateClique, adjacentVertices, incumbentVertices, globalQueue, populate);
	}
}


int main(){
	globalQueue = (struct returnObject*)malloc(sizeof(struct returnObject)*COUNT);
	int *incumbentVertices = (int *)malloc(sizeof(int)*COUNT);
	int *candidateClique = (int *)malloc(sizeof(int)*COUNT);
	int *adjacentVertices = (int *)malloc(sizeof(int)*COUNT);
	int graph1[ROWS][2] = { { 4, 5 }, { 2, 5 }, { 5, 6 }, { 3, 5 }, { 2, 4 }, { 3, 4 }, { 2, 3 }, { 0, 1 }, { 0, 2 }, { 0, 6 }, { 1, 2 }, { 1, 6 } };
	struct vertex *graph = (struct vertex*)malloc(sizeof(struct vertex)*ROWS);
	int i;
	for (i = 0; i<ROWS; i++){
		graph[i].x = graph1[i][0];
		graph[i].y = graph1[i][1];
	}
	int populate = 1;
	int *cudaIncumbent, *cudaCandidate, *cudaAdjacent;
	struct vertex *cudaGraph;
	int sizeVertices = sizeof(int)*COUNT;
	int sizeGraph = sizeof(struct vertex)*ROWS;
	hipMalloc((void **)&cudaIncumbent, sizeVertices);
	hipMalloc((void **)&cudaCandidate, sizeVertices);
	hipMalloc((void **)&cudaAdjacent, sizeVertices);
	hipMalloc((void **)&cudaGraph, sizeGraph);

	for (i = 0; i<COUNT; i++){
		incumbentVertices[i] = -1;
		candidateClique[i] = -1;
		adjacentVertices[i] = -1;
	}
	for (i = 0; i < COUNT; ++i){
		adjacentVertices[i] = 1;
	}


	hipMemcpy(cudaIncumbent, incumbentVertices, sizeVertices, hipMemcpyHostToDevice);
	hipMemcpy(cudaAdjacent, adjacentVertices, sizeVertices, hipMemcpyHostToDevice);
	hipMemcpy(cudaCandidate, candidateClique, sizeVertices, hipMemcpyHostToDevice);
	hipMemcpy(cudaGraph, graph, sizeGraph, hipMemcpyHostToDevice);

	clock_t t;
	t = clock();
	expand << <1, 1 >> >(graph, candidateClique, adjacentVertices, incumbentVertices,globalQueue,populate);
	//if (queueFront <= queueRear)
	populate = 0;
		expand << <findQueueSiz() + 1, 1 >> >(graph, candidateClique, adjacentVertices, incumbentVertices, globalQueue,populate);

	hipError_t e = hipMemcpy(incumbentVertices, cudaIncumbent, sizeVertices, hipMemcpyDeviceToHost);
	//if (e != hipSuccess)
		//printf("%s", hipGetErrorString(e));

	//	int incumbentSize=findSize(incumbentVertices);
	expand(graph1, candidateClique, adjacentVertices, incumbentVertices);
	printf("The maximum clique constitutes the vertices: ");
	for (i = 0; i<COUNT; i++){
		if (incumbentVertices[i] == 1)
			printf("%d   ", i + 1);
	}
	printf("\n");
	t = clock() - t;
	double time_taken = ((double)t) / CLOCKS_PER_SEC;
	printf("\nThe algorithm took %f seconds to find the maximum clique of the given graph!\n", time_taken);
	return 0;
}
